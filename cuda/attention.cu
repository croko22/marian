
#include <hip/hip_runtime.h>
#include "attention.cuh"
#include "matrix_utils.cuh"

void run_attention(float *h_Q, float *h_K, float *h_V, float *h_output, int N)
{
    float *d_Q, *d_K, *d_V, *d_score, *d_out;
    size_t size = N * N * sizeof(float);

    hipMalloc(&d_Q, size);
    hipMalloc(&d_K, size);
    hipMalloc(&d_V, size);
    hipMalloc(&d_score, size);
    hipMalloc(&d_out, size);

    hipMemcpy(d_Q, h_Q, size, hipMemcpyHostToDevice);
    hipMemcpy(d_K, h_K, size, hipMemcpyHostToDevice);
    hipMemcpy(d_V, h_V, size, hipMemcpyHostToDevice);

    dim3 threads(16, 16);
    dim3 blocks((N + 15) / 16, (N + 15) / 16);

    matmul<<<blocks, threads>>>(d_Q, d_K, d_score, N);
    row_softmax<<<N, 1>>>(d_score, N);
    matmul<<<blocks, threads>>>(d_score, d_V, d_out, N);

    hipMemcpy(h_output, d_out, size, hipMemcpyDeviceToHost);

    hipFree(d_Q);
    hipFree(d_K);
    hipFree(d_V);
    hipFree(d_score);
    hipFree(d_out);
}
